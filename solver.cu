#include "hip/hip_runtime.h"
#include "FDTD2D.h"

#define sigma 10

//__device__ double
//Hsource(int type, double t){
//	switch(type){
//	case Sin:
//		return (t>0)?atan(t/20)*sin((t*1e-2*2*3.14)):0;
//		break;
//	case Pulse:
//		return (t>0)?exp(-0.5*(t-3*sigma)*(t-3*sigma)/(sigma*sigma)):0;
//		break;
//	case None:
//		return 0;
//		break;
//	}
//}

//H
__global__ void
updateH(mesh* m, int W, int H)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int k = idy*W+idx;
	//update
	if(idx<W-1 && idx > 1 && idy > 1 && idy<H-1){
		m[k].Hx=m[k].Dax*m[k].Hx-m[k].Dbx*(m[k+W].Ey-m[k].Ey);
		m[k].Hy=m[k].Day*m[k].Hy+m[k].Dby*(m[k+1].Ex-m[k].Ex);
		m[k].Hz = m[k].Hx+m[k].Hy;
	}
}

//E
__global__ void
updateE(mesh *m, int W, int H)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int k = idy*W+idx;
	//update
	if(idx<W-1 && idx > 1 && idy > 1 && idy<H-1){
		m[k].Ex=m[k].Cax*m[k].Ex+m[k].Cbx*(m[k].Hz-m[k-1].Hz);
		m[k].Ey=m[k].Cay*m[k].Ey-m[k].Cby*(m[k].Hz-m[k-W].Hz);
	}
}

__global__ void 
updateSource(mesh *m, int W, int H){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int k = idy*W+idx;
	if(idx<W-1 && idx > 1 && idy > 1 && idy<H-1){
		m[k].AccumSource += Hsource(m[k].Srctype, m[k].sourceTimer);
		m[k].Hz+=m[k].AccumSource;
		m[k].sourceTimer=(m[k].Srctype==None)?0:m[k].sourceTimer+1;
	}
}

#define BLOCKSIZ 32

extern "C"
void cudaUpdateKernel(mesh* d_m, int Nx, int Ny){
	dim3 dimBlock(BLOCKSIZ,BLOCKSIZ);
	dim3 dimGrid(ceil(Nx/BLOCKSIZ), ceil(Ny/BLOCKSIZ));
	updateH<<<dimGrid, dimBlock>>>(d_m, Nx, Ny);
	updateSource<<<dimGrid, dimBlock>>>(d_m, Nx, Ny);
	updateE<<<dimGrid, dimBlock>>>(d_m, Nx, Ny);
}

