#include "hip/hip_runtime.h"
#include "FDTD2D.h"

//__device__ double
//source(int i, int j, double t){
//    return (j==200)?8*sin((t*1e10*2*pi)):0;
//}

//H
__global__ void
updateH(mesh* m, int W, int H)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int k = idy*W+idx;
	//update
	if(idx<W-1 && idx > 1 && idy > 1 && idy<H-1){
		m[k].Hx=m[k].Dax*m[k].Hx-m[k].Dbx*(m[k+W].Ey-m[k].Ey);
		m[k].Hy=m[k].Day*m[k].Hy+m[k].Dby*(m[k+1].Ex-m[k].Ex);
	}
}

//E
__global__ void
updateE(mesh *m, int W, int H)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int k = idy*W+idx;
	//update
	if(idx<W-1 && idx > 1 && idy > 1 && idy<H-1){
		m[k].Ex=m[k].Cax*m[k].Ex+m[k].Cbx*(m[k].Hz-m[k-1].Hz);
		m[k].Ey=m[k].Cay*m[k].Ey-m[k].Cby*(m[k].Hz-m[k-W].Hz);
	}
}

__global__ void 
updateSource(mesh *m, int W, int H, double time){
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	int idy = blockIdx.y * blockDim.y + threadIdx.y;
//	int k = idy*W+idx;
//	if(idx<W && idx>0 && idy>0 && idy<H){
//		double s = source(idx, idy, time);
//		//double Hs = source(idx, idy, time);
//		//double Es = source(idx, idy, time);
//		m[k].Hzy+= m[k].DHx2*	s;
//		m[k+1].Ey	+= m[k+1].CEy2*s/(120*pi);
//	}
}

#define BLOCKSIZ 128

extern "C"
void cudaUpdateKernel(mesh* d_m, int Nx, int Ny, double t){
	dim3 dimBlock(BLOCKSIZ,BLOCKSIZ);
	dim3 dimGrid(ceil(Nx/BLOCKSIZ), ceil(Ny/BLOCKSIZ));
	updateH<<<dimGrid, dimBlock>>>(d_m, Nx, Ny);
	updateE<<<dimGrid, dimBlock>>>(d_m, Nx, Ny);
	updateSource<<<dimGrid, dimBlock>>>(d_m, Nx, Ny, t);
}

