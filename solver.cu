#include "hip/hip_runtime.h"
#include "FDTD2D.h"
#include <stdio.h>

#define sigma 10

__device__ double
Hsource(mesh* m){
    //double sigma=10;
	double t = m->sourceTimer;
	switch(m->Srctype){
	case Sin:
		return sin((t*1e-2*2*3.14));
		break;
	case Pulse: 
		return exp(-0.5*(t-3*sigma)*(t-3*sigma)/(sigma*sigma));
		break;
	case None:
		return 0;
		break;
	}
}
__device__ void 
source(mesh *m, double a){
    //m->AccumSource+=Hsource(m, m->sourceTimer);
    m->AccumSource+=a;
    //m->Hz-=m->AccumSource;
    m->Hz=10;
    m->sourceTimer=(m->Srctype==None)?0:m->sourceTimer+1;
}

//H
__global__ void
updateH(mesh* m, int W, int H)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int k = idy*W+idx;
	//update
	if(idx<W-1 && idx > 1 && idy > 1 && idy<H-1){
		m[k].Hx=m[k].Dax*m[k].Hx-m[k].Dbx*(m[k+W].Ey-m[k].Ey);
		m[k].Hy=m[k].Day*m[k].Hy+m[k].Dby*(m[k+1].Ex-m[k].Ex);
		m[k].Hz = m[k].Hx+m[k].Hy;
	}
}

//E
__global__ void
updateE(mesh *m, int W, int H)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int k = idy*W+idx;
	//update
	if(idx<W-1 && idx > 1 && idy > 1 && idy<H-1){
		m[k].Ex=m[k].Cax*m[k].Ex+m[k].Cbx*(m[k].Hz-m[k-1].Hz);
		m[k].Ey=m[k].Cay*m[k].Ey-m[k].Cby*(m[k].Hz-m[k-W].Hz);
	}
}

__global__ void 
updateSource(mesh *m, int W, int H){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int k = idy*W+idx;
	if(idx<W-1 && idx > 1 && idy > 1 && idy<H-1){
		m[k].Hz = 100;
		//double a = Hsource(&m[k]);
		//source(&m[k], a);
	}
}

#define BLOCKSIZ 128

extern "C"
void cudaUpdateKernel(mesh* d_m, int Nx, int Ny){
	dim3 dimBlock(BLOCKSIZ,BLOCKSIZ);
	dim3 dimGrid(ceil(Nx/BLOCKSIZ), ceil(Ny/BLOCKSIZ));
	updateH<<<dimGrid, dimBlock>>>(d_m, Nx, Ny);
	updateE<<<dimGrid, dimBlock>>>(d_m, Nx, Ny);
	updateSource<<<dimGrid, dimBlock>>>(d_m, Nx, Ny);
}

